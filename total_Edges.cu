
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 4
//no of nodes

__device__ int total_edges=0;

__global__ void kernel(int * adjmatrix){

    int row=blockIdx.x*blockDim.x+threadIdx.x;
    if(row<N){
    int sum=0;
    for(int i=0;i<N;++i){
        sum+=adjmatrix[row*N+i];
    }
    printf("sum: %d ",sum);
    atomicAdd(&total_edges,sum);
  }
}

int main(){
    int *gpuadjmatrix;
    int cpuadjmatrix[4][4]={{0,1,1,0},{0,0,1,1},{1,1,0,1},{0,0,0,0}};
    int ans;
    hipMalloc((void**)&gpuadjmatrix,N*N*sizeof(int));
    hipMemcpy(gpuadjmatrix,cpuadjmatrix,N*N*sizeof(int),hipMemcpyHostToDevice);
    kernel<<<1,N>>>(gpuadjmatrix);
hipMemcpyFromSymbol(&ans, HIP_SYMBOL(total_edges), sizeof(int));
    printf("Total number of edges: %d\n", ans);
    hipFree(gpuadjmatrix);

    return 0;

}