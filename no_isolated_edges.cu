

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 4

__device__ int no_of_isolated_nodes=0;
__device__ int isolated_nodes[N];

__global__ void kernel(int* adjmatrix){
int row=blockIdx.x*blockDim.x + threadIdx.x;
if(row<N){
int count=0;

for(int col=0;col<N;++col){
if(adjmatrix[row*N+ col]==0) count++;
}
if(count==N){ 
int index = atomicAdd(&no_of_isolated_nodes, 1);
isolated_nodes[index] = row;
}
}
}

int main(){
  int cpumatrix[4][4]={{0,1,1,0},{0,0,0,0},{1,1,1,0},{0,0,0,0}};
  int* gpumatrix;
  hipMalloc(&gpumatrix,N*N*sizeof(int));
  hipMemcpy(gpumatrix,cpumatrix,N*N*sizeof(int),hipMemcpyHostToDevice);
  kernel<<<1,N>>>(gpumatrix);

  int no_of_isolated_nodes_cpu;
  int isolated_nodes_cpu[N];
  hipMemcpyFromSymbol(&no_of_isolated_nodes_cpu,HIP_SYMBOL(no_of_isolated_nodes),sizeof(int));
  hipMemcpyFromSymbol(&isolated_nodes_cpu,HIP_SYMBOL(isolated_nodes),N*sizeof(int));

  printf(" No of isolated nodes are: %d \n",no_of_isolated_nodes_cpu);
  printf("The isolated nodes are: \n");
  for(int i=0;i<no_of_isolated_nodes_cpu;++i){
    printf("%d ",isolated_nodes_cpu[i]);
  }
}