#include "hip/hip_runtime.h"
%%writefile bfs.cu

#include <stdio.h>
#define N 7

__global__ void kernel(int* visited,int* layer,int layer_size,int* g_next_size,int* g_next_layer,int* level,int* curr_level,int *matrix){
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  if(id<layer_size){
    int u=layer[id];
    for(int i=0;i<N;++i){
       if (matrix[u * N + i] == 1 && visited[i] == 0) {
        if (atomicExch(&visited[i], 1) == 0) {
        int pos=atomicAdd(g_next_size,1);
        g_next_layer[pos]=i;
        level[i]=*curr_level+1;
      }
    }
  }
}
}

int main() {
    // Adjacency matrix of the graph
    int adj_matrix[N][N] = {
        {0, 1, 0, 1, 0, 0, 0},
        {1, 0, 1, 0, 1, 0, 0},
        {0, 1, 0, 0, 0, 0, 0},
        {1, 0, 0, 0, 1, 1, 0},
        {0, 1, 0, 1, 0, 0, 0},
        {0, 0, 0, 1, 0, 0, 1},
        {0, 0, 0, 0, 0, 1, 0}
    };
int* gpumatrix;
hipMalloc(&gpumatrix,N*N*sizeof(int));
hipMemcpy(gpumatrix,adj_matrix,N*N*sizeof(int),hipMemcpyHostToDevice);

int* g_visited;
int visited[N]={0};
visited[0]=1;
hipMalloc(&g_visited,N*sizeof(int));
hipMemcpy(g_visited,visited,N*sizeof(int),hipMemcpyHostToDevice);


int layer[N];
layer[0]=0;
visited[0]=1;
int layer_size=1;

int* g_layer;
hipMalloc(&g_layer,N*sizeof(int));
hipMemcpy(g_layer,layer,N*sizeof(int),hipMemcpyHostToDevice);

/*int next_layer[N];*/
int* g_next_layer;
hipMalloc(&g_next_layer,N*sizeof(int));

//int next_layer_size;
int* g_next_layer_size;
hipMalloc(&g_next_layer_size,sizeof(int));

int level[N]={0};
int* g_level;
hipMalloc(&g_level,N*sizeof(int));
hipMemcpy(g_level,level,N*sizeof(int),hipMemcpyHostToDevice);

int curr_level=0;
int* g_curr_level;
hipMalloc(&g_curr_level,sizeof(int));
hipMemcpy(g_curr_level,&curr_level,sizeof(int),hipMemcpyHostToDevice);

while(layer_size>0){

kernel<<<1, N>>>(g_visited, g_layer, layer_size, g_next_layer_size, g_next_layer, g_level, g_curr_level, gpumatrix);
hipDeviceSynchronize(); 
hipMemcpy(&curr_level,g_curr_level,sizeof(int),hipMemcpyDeviceToHost);
curr_level++;
hipMemcpy(g_curr_level,&curr_level,sizeof(int),hipMemcpyHostToDevice);

//now next_layer becomes layer

hipMemcpy(layer, g_next_layer, N * sizeof(int), hipMemcpyDeviceToHost);
hipMemcpy(g_layer, g_next_layer, N * sizeof(int), hipMemcpyDeviceToDevice);
hipMemcpy(&layer_size, g_next_layer_size, sizeof(int), hipMemcpyDeviceToHost);
int zero = 0;
//next_layer_size = 0;
hipMemcpy(g_next_layer_size, &zero, sizeof(int), hipMemcpyHostToDevice);

}

hipMemcpy(level,g_level,N*sizeof(int),hipMemcpyDeviceToHost);

for(int i=0;i<N;++i){
  printf("Node %d: Level :%d\n",i,level[i]);
}

//cleanup

hipFree(gpumatrix);
hipFree(g_visited);
hipFree(g_layer);
hipFree(g_next_layer);
hipFree(g_next_layer_size);
hipFree(g_level);
hipFree(g_curr_level);

}
