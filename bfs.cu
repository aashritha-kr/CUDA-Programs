#include "hip/hip_runtime.h"
%%writefile bfs.cu

#include <stdio.h>
#define N 7

__global__ void kernel(int* visited,int* frontier,int frontier_size,int* g_next_size,int* g_next_frontier,int* level,int* curr_level,int *matrix){
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  if(id<frontier_size){
    int u=frontier[id];
    for(int i=0;i<N;++i){
       if (matrix[u * N + i] == 1 && visited[i] == 0) {
        if (atomicExch(&visited[i], 1) == 0) {
        int pos=atomicAdd(g_next_size,1);
        g_next_frontier[pos]=i;
        level[i]=*curr_level+1;
      }
    }
  }
}
}

int main() {
    // Adjacency matrix of the graph
    int adj_matrix[N][N] = {
        {0, 1, 0, 1, 0, 0, 0},
        {1, 0, 1, 0, 1, 0, 0},
        {0, 1, 0, 0, 0, 0, 0},
        {1, 0, 0, 0, 1, 1, 0},
        {0, 1, 0, 1, 0, 0, 0},
        {0, 0, 0, 1, 0, 0, 1},
        {0, 0, 0, 0, 0, 1, 0}
    };
int* gpumatrix;
hipMalloc(&gpumatrix,N*N*sizeof(int));
hipMemcpy(gpumatrix,adj_matrix,N*N*sizeof(int),hipMemcpyHostToDevice);

int* g_visited;
int visited[N]={0};
visited[0]=1;
hipMalloc(&g_visited,N*sizeof(int));
hipMemcpy(g_visited,visited,N*sizeof(int),hipMemcpyHostToDevice);


int frontier[N];
frontier[0]=0;
visited[0]=1;
int frontier_size=1;

int* g_frontier;
hipMalloc(&g_frontier,N*sizeof(int));
hipMemcpy(g_frontier,frontier,N*sizeof(int),hipMemcpyHostToDevice);

/*int next_frontier[N];*/
int* g_next_frontier;
hipMalloc(&g_next_frontier,N*sizeof(int));

//int next_frontier_size;
int* g_next_frontier_size;
hipMalloc(&g_next_frontier_size,sizeof(int));

int level[N]={0};
int* g_level;
hipMalloc(&g_level,N*sizeof(int));
hipMemcpy(g_level,level,N*sizeof(int),hipMemcpyHostToDevice);

int curr_level=0;
int* g_curr_level;
hipMalloc(&g_curr_level,sizeof(int));
hipMemcpy(g_curr_level,&curr_level,sizeof(int),hipMemcpyHostToDevice);

while(frontier_size>0){

kernel<<<1, N>>>(g_visited, g_frontier, frontier_size, g_next_frontier_size, g_next_frontier, g_level, g_curr_level, gpumatrix);
hipDeviceSynchronize(); 
hipMemcpy(&curr_level,g_curr_level,sizeof(int),hipMemcpyDeviceToHost);
curr_level++;
hipMemcpy(g_curr_level,&curr_level,sizeof(int),hipMemcpyHostToDevice);

//now next_frontier becomes frontier

hipMemcpy(frontier, g_next_frontier, N * sizeof(int), hipMemcpyDeviceToHost);
hipMemcpy(g_frontier, g_next_frontier, N * sizeof(int), hipMemcpyDeviceToDevice);
hipMemcpy(&frontier_size, g_next_frontier_size, sizeof(int), hipMemcpyDeviceToHost);
int zero = 0;
//next_frontier_size = 0;
hipMemcpy(g_next_frontier_size, &zero, sizeof(int), hipMemcpyHostToDevice);

}

hipMemcpy(level,g_level,N*sizeof(int),hipMemcpyDeviceToHost);

for(int i=0;i<N;++i){
  printf("Node %d: Level :%d\n",i,level[i]);
}

//cleanup

hipFree(gpumatrix);
hipFree(g_visited);
hipFree(g_frontier);
hipFree(g_next_frontier);
hipFree(g_next_frontier_size);
hipFree(g_level);
hipFree(g_curr_level);

}
