#include "hip/hip_runtime.h"
%%writefile matrixadd.cu
#include <stdio.h>

#define N 5
#define M 6
__global__ void addition(int* gpumatrix){
unsigned id=threadIdx.y * blockDim.x + threadIdx.x;
gpumatrix[id]=id;
}
int main(){
  int* gpumatrix,*cpumatrix;
  dim3 blocks(N,M,1);
  hipMalloc((void**)&gpumatrix,N*M*sizeof(int));
  addition<<<1,blocks>>>(gpumatrix);
  cpumatrix=(int*)(malloc)(N*M*sizeof(int));

  hipMemcpy(cpumatrix,gpumatrix,N*M*sizeof(int),hipMemcpyDeviceToHost);
  for(int i=0;i<N;++i){
    for(int j=0;j<M;++j){
      printf("%d ",cpumatrix[i*M+j]);
    }
    printf("\n");
  }

}