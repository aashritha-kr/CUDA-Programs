#include "hip/hip_runtime.h"
%%writefile matrixadd.cu
#include <stdio.h>

#define N 5
#define M 6

__global__ void init(int* matrix1,int* matrix2){
unsigned id=threadIdx.y * blockDim.x + threadIdx.x;
matrix1[id]=id;
matrix2[id]=2*id;
}

__global__ void addition(int* matrix1,int* matrix2,int* result){
unsigned id=threadIdx.y * blockDim.x + threadIdx.x;
result[id]=matrix1[id]+matrix2[id];
}
int main(){
  int* gpumatrix1,*gpumatrix2,*resultmatrix;
  int* cpumatrix;

  hipMalloc((void**)&gpumatrix1,N*M*sizeof(int));
  hipMalloc((void**)&gpumatrix2,N*M*sizeof(int));
  hipMalloc((void**)&resultmatrix,N*M*sizeof(int));
  
  cpumatrix=(int*)malloc(N*M*sizeof(int));
  
  dim3 block(N,M,1);

  init<<<1,block>>> (gpumatrix1,gpumatrix2);
  addition<<<1,block>>> (gpumatrix1,gpumatrix2,resultmatrix);

  hipMemcpy(cpumatrix,resultmatrix,N*M*sizeof(int),hipMemcpyDeviceToHost);

  for(int i=0;i<N;++i){
    for(int j=0;j<M;++j){
      printf("%d ",cpumatrix[i*M+j]);
    }
    printf("\n");
  }

  hipFree(gpumatrix1);
  hipFree(gpumatrix2);
  hipFree(resultmatrix);
  free(cpumatrix);

}