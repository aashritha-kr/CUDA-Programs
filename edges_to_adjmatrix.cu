#include "hip/hip_runtime.h"
%%writefile edge_to_adj_matrix.cu
#define E 6
#define N 4

__global__ void kernel(int* matrix,int* edges){
int id=blockIdx.x*blockDim.x+threadIdx.x;
if(id<E){
  int u=edges[id*2+0];
  int v=edges[id*2+1];
  matrix[u*N+v]=1;
  matrix[v*N+u]=1;
}

}

#include <stdio.h>
int main(){
      int edges[E][2] = {{0,1}, {0,2}, {1,2}, {2,0}, {2,3}, {3,3}};
      int adjmatrix[N][N]={0};

      int* gpumatrix;
      int* gpuedges;

      hipMalloc(&gpumatrix,N*N*sizeof(int));
      hipMalloc(&gpuedges,E*2*sizeof(int));

      hipMemcpy(gpumatrix,adjmatrix,N*N*sizeof(int),hipMemcpyHostToDevice);
      hipMemcpy(gpuedges,edges,E*2*sizeof(int),hipMemcpyHostToDevice);

      kernel<<<1,E>>> (gpumatrix,gpuedges);

      hipMemcpy(adjmatrix,gpumatrix,N*N*sizeof(int),hipMemcpyDeviceToHost);

      for(int i=0;i<N;++i){
        for(int j=0;j<N;++j){
          printf("%d ",adjmatrix[i][j]);
        }
        printf("\n");
      }

      hipFree(gpumatrix);
      hipFree(gpuedges);

}