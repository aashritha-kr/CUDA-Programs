#include "hip/hip_runtime.h"
%%writefile symmetric.cu

#include <stdio.h>
#define N 18

__device__ int symmetric=0;

__global__ void kernel(int* adjmatrix){

int id=blockIdx.x*blockDim.x+ threadIdx.x;

int row=id/N;
int col=id%N;

if(col>row && adjmatrix[row*N+col]==adjmatrix[col*N+row]){
atomicAdd(&symmetric,1);
}
}

int main(){
  int cpumatrix[N][N]={
    {0,1,0,0,0,0,0,0,1,0,0,0,0,0,0,0,0,0}, 
    {1,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0}, 
    {0,1,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0,0}, 
    {0,0,1,0,1,0,0,0,0,0,0,0,0,0,0,0,0,0}, 
    {0,0,0,1,0,1,0,0,0,0,0,0,0,0,0,0,0,0}, 
    {0,0,0,0,1,0,1,0,0,0,0,0,0,0,0,0,0,0}, 
    {0,0,0,0,0,1,0,1,0,0,0,0,0,0,0,0,0,0}, 
    {0,0,0,0,0,0,1,0,1,0,0,0,0,0,0,0,0,0}, 
    {1,0,0,0,0,0,0,1,0,1,0,0,0,0,0,0,0,0}, 
    {0,0,0,0,0,0,0,0,1,0,1,0,0,0,0,0,0,0}, 
    {0,0,0,0,0,0,0,0,0,1,0,1,0,0,0,0,0,0}, 
    {0,0,0,0,0,0,0,0,0,0,1,0,1,0,0,0,0,0}, 
    {0,0,0,0,0,0,0,0,0,0,0,1,0,1,0,0,0,0}, 
    {0,0,0,0,0,0,0,0,0,0,0,0,1,0,1,0,0,0}, 
    {0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,1,0,0}, 
    {0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,1,0}, 
    {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,1,0,1}, 
    {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0}};

    int* gpumatrix;
    hipMalloc(&gpumatrix,N*N*sizeof(int));
    hipMemcpy(gpumatrix,cpumatrix,N*N*sizeof(int),hipMemcpyHostToDevice);
    int blocksize=16;
    int gridsize=(N*N+blocksize-1)/blocksize;
    kernel<<<gridsize,blocksize>>>(gpumatrix);
    int ans;
    hipMemcpyFromSymbol(&ans,HIP_SYMBOL(symmetric),sizeof(int));
    if(ans==(N*(N-1)/2)) printf("This is a symmetric graph\n");

    printf("%d",ans);

}