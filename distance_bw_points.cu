#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void computeDistances(int *d_x, int *d_y, int num_points,float *d_distances)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_points * num_points)
    {
        int i = idx / num_points;
        int j = idx % num_points;
        if (i != j)
        {
            // Skip distance from a point to itself
            int dx = d_x[j] - d_x[i];
            int dy = d_y[j] - d_y[i];
            d_distances[idx] = sqrtf(dx * dx + dy * dy);
        }
        else
        {
            d_distances[idx] = 0.0f; // Distance to itself is zero
        }
    }
}

int main()
{
    const char *file1 = "x.txt";
    const char *file2 = "y.txt";
    vector<int> x, y;
    // Read x coordinates
    ifstream infile1(file1);
    int num;
    while (infile1 >> num)
    {
        x.push_back(num);
    }
    infile1.close();
    // Read y coordinates
    ifstream infile2(file2);
    while (infile2 >> num)
    {
        y.push_back(num);
    }
    infile2.close();

    int num_points = x.size();
    if (num_points != y.size() || num_points < 2)
    {
        cerr << "Invalid input data." << endl;
        return -1;
    }
    // Allocate memory for x, y, distances on the device
    int *d_x, *d_y;
    float *d_distances;
    int blockSize = 256;
    int numBlocksDistance = (num_points * num_points + blockSize - 1) /
                            blockSize;
    computeDistances<<<numBlocksDistance, blockSize>>>(d_x, d_y,
                                                       num_points, d_distances);

    // Copy distances from device to host
    vector<float> distances(num_points * num_points);


// Print distances
for (int i = 0; i < num_points; ++i) {
        for (int j = 0; j < num_points; ++j)
        {
            cout << distances[i * num_points + j] << " ";
        }
        cout << endl;}

// Free device memory
hipFree(d_x);
hipFree(d_y);
hipFree(d_distances);
return 0;
}