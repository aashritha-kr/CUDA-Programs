#include "hip/hip_runtime.h"
%%writefile self_loops.cu
#include <stdio.h>
#define N 4

__device__ int no_of_self_loops=0;

__global__ void kernel(int* adjmatrix){
  int i=blockIdx.x*blockDim.x+threadIdx.x;
  if(id<N){  if(adjmatrix[i*N+i]==1) atomicAdd(&no_of_self_loops,1);}
}


int main(){
  int cpumatrix[N][N]={{1,0,0,0},{1,1,0,0},{1,0,0,0},{1,0,0,0}};
  int* gpumatrix;
  hipMalloc(&gpumatrix,N*N*sizeof(int));
  hipMemcpy(gpumatrix,cpumatrix,N*N*sizeof(int),hipMemcpyHostToDevice);
  kernel<<<(N+2)/3,3>>>(gpumatrix);
  int ans;
hipMemcpyFromSymbol(&ans,HIP_SYMBOL(no_of_self_loops),sizeof(int));
printf("%d",ans);
hipFree(gpumatrix);
}
